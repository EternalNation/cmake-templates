/**
 * Author: Nikolaus Mayer, 2014  (mayern@informatik.uni-freiburg.de)
 * CUDA kernels
 */


#include <hip/hip_runtime.h>
#include <cmath>  // std::ceil

/**
 * Kernel
 */
__global__ void generic_CUDA_function__kernel( float* DATA,
                                               size_t data_size
                                             )
{
  const size_t x = blockIdx.x * blockDim.x + threadIdx.x;
  if ( x >= data_size )
    return;

  DATA[x] *= 2.0f;  
}


/**
 * Function
 */
void generic_CUDA_function( float* DATA,
                            size_t data_size
                          )
{
  /// CUDA kernel parameters
  const dim3 block(16, 1, 1);
  const dim3 grid(std::ceil(data_size/(float)block.x), 1, 1);

  /// Call kernel
  generic_CUDA_function__kernel<<<grid,block>>>( DATA, data_size );
}

